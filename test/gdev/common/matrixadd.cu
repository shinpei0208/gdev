#include <stdint.h>
#include <hip/hip_runtime.h>
extern "C" __global__ void bench(uint32_t *a, uint32_t *b, uint32_t *c, uint32_t n){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if(i<n&&j<n){
        int idx = i*n+j;
        c[idx] = a[idx] + b[idx];
    }
}
